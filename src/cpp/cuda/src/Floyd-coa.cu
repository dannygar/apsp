#include "hip/hip_runtime.h"
/**
 * Copyright 2019  Microsoft Corporation.  All rights reserved.
 *
 * Please refer to the Microsoft end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 * Floyd-Warshall with coalesced memory optimization
 */
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""

#include "cuda/inc/Floyd.cuh"
#include "inc/test.h"


void Floyd_Warshall_COA(int *matrix, int* path, unsigned int size, float* time)
{
	hipEvent_t start, stop;

	// Initialize CUDA GPU Timers
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Start CUDA Timer
	hipEventRecord(start, nullptr);

	// allocate memory
	int *matrixOnGPU;
	int *pathOnGPU;
	hipMalloc(reinterpret_cast<void **>(&matrixOnGPU), sizeof(int)*size*size);
	hipMemcpy(matrixOnGPU, matrix, sizeof(int)*size*size, hipMemcpyHostToDevice);
	hipMalloc(reinterpret_cast<void **>(&pathOnGPU), sizeof(int)*size*size);
	hipMemcpy(pathOnGPU, path, sizeof(int)*size*size, hipMemcpyHostToDevice);

	// dimension
	dim3 dimGrid(size / COA_TILE_WIDTH, size / COA_TILE_WIDTH, 1);
	dim3 dimBlock(COA_TILE_WIDTH, COA_TILE_WIDTH, 1);

	// run kernel
	for (unsigned int k = 0; k < size; ++k)
		cudaKernel_coa <<< dimGrid, dimBlock >>> (matrixOnGPU, pathOnGPU, size, k);

	// get result back
	hipMemcpy(matrix, matrixOnGPU, sizeof(int)*size*size, hipMemcpyDeviceToHost);
	hipMemcpy(path, pathOnGPU, sizeof(int)*size*size, hipMemcpyDeviceToHost);

	// Stop CUDA Timer
	hipEventRecord(stop, nullptr);
	//Synchronize GPU with CPU
	hipEventSynchronize(stop);

	// Read the elapsed time and release memory
	hipEventElapsedTime(*&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);


	// free memory resources
	hipFree(matrixOnGPU);
	hipFree(pathOnGPU);
}


__global__ void cudaKernel_coa(int *matrix, int* path, int size, int k)
{
	// compute indexes
	const int v = blockDim.y * blockIdx.y + threadIdx.y;
	const int  u = blockDim.x * blockIdx.x + threadIdx.x;

	const int  i0 = v * size + u;
	const int  i1 = v * size + k;
	const int  i2 = k * size + u;

	// read in dependent values
	const int  i0_value = matrix[i0];
	const int  i1_value = matrix[i1];
	const int  i2_value = matrix[i2];


	// Synchronize to make sure that all value are current
	__syncthreads();

	// calculate Floyd-Warshall shortest path
	if (i1_value != INF && i2_value != INF)
	{
		const int sum = i1_value + i2_value;
		if (i0_value == INF || sum < i0_value)
		{
			matrix[i0] = sum;
			path[i0] = path[i2];
		}
	}
}

